#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

//#define DEBUG

//Variablen
#define p 3
#define q 5
#define n 15
#define e 3
#define v 3
#define z 8
#define anzahl_Texte 1000
#define count_cores 8

//__device__ long int klartexte[anzahl_Texte];
//__device__ long int klartexte_pruefung[anzahl_Texte];
//__device__ long int geheimtexte[anzahl_Texte];

/*
Klartext: K
Geheimtext: G
Verschluesselung: G = K^v mod n
Entschluesselung: K = G^e mod n

Index des CUDA Kerns: blockIdx.x blockIdx.y

*/

static void HandleError( hipError_t err, const char *file, int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void verschluessselung(int klartexte[], int geheimtexte[])
{
	int i, j, multi, x;
	
	int block_length = anzahl_Texte/count_cores;
	
	for (i = 0 ; i < block_length; i++)
	{
		//Integer hoch 103 ist zu hoch!
		//geheimtexte[i+blockIdx.x*block_length] = pow(klartexte[i+blockIdx.x*block_length],3) % 15;
		//geheimtexte[i+blockIdx.x*block_length] = mypow(,v);
		
		multi = x  = klartexte[i+blockIdx.x*block_length];
		for (j = 1; i < v; i++)
			x *= multi;
		
		geheimtexte[i+blockIdx.x*block_length] = x % n;
	}
}


__global__ void entschluessselung(int geheimtexte[], int klartexte_pruefung[])
{
	int i;
	
	int block_length = anzahl_Texte/count_cores;

	for (i = 0 ; i < block_length; i++)
	{
		//Integer hoch 103 ist zu hoch!
		//klartexte_pruefung[i+blockIdx.x*block_length] = pow(geheimtexte[i+blockIdx.x*block_length],e) % n;
	}
}

__device__ int mypow(int x, int y)
{
	
	int i;
	int multi = x;
	for (i = 1; i < y; i++)
	{
		x *= multi;
	}
	return x;
}

int main(void) {
	int i, j;
	hipEvent_t start, stop;
	float elapsedTime;
	
	int klartexte[anzahl_Texte];
	int klartexte_pruefung[anzahl_Texte];
	int geheimtexte[anzahl_Texte];

	//Klartetexte Array belegen
	//rand initialisieren
	srand((unsigned)time(NULL));
	for (i = 0; i < anzahl_Texte; i ++)
	{
		klartexte[i] = rand() % 10;		//Zahlen nicht  zu gross waehlen
	}

	int *dev_klartexte, *dev_geheimtexte, *dev_klartexte_pruefung;

	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));



	HANDLE_ERROR(hipEventRecord(start, 0));

        HANDLE_ERROR(hipMalloc((void **)&dev_klartexte, sizeof(klartexte)));
        HANDLE_ERROR(hipMalloc((void **)&dev_geheimtexte, sizeof(geheimtexte)));
        HANDLE_ERROR(hipMalloc((void **)&dev_klartexte_pruefung, sizeof(klartexte_pruefung)));

        HANDLE_ERROR(hipMemcpy(dev_klartexte, klartexte, sizeof(klartexte), hipMemcpyHostToDevice));
        //HANDLE_ERROR(hipMemcpy(dev_matN, matN, sizeof(matN), hipMemcpyHostToDevice));

	dim3 blocks(count_cores, 1);

	verschluessselung<<<blocks, 1>>>(dev_klartexte, dev_geheimtexte);

        HANDLE_ERROR(hipMemcpy(geheimtexte, dev_geheimtexte, sizeof(geheimtexte), hipMemcpyDeviceToHost));
		
		printf("Die Geheimtexte wurden verschluesselt.\n\nGeheimtexte:\n");
		for (i = 0; i < anzahl_Texte; i++)
		{
			printf("%ld, ", geheimtexte[i]);
		}
		printf("\n\n");
		

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));

	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("Elapsed time: %3.1f ms\n", elapsedTime);



	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));
	

	return EXIT_SUCCESS;
}
